#include "hip/hip_runtime.h"
#ifndef _CUDA_KERNEL_H_
#define _CUDA_KERNEL_H_

#include "globals.h"

__device__ float* g_data[2];
__device__ int maximum_index;
__device__ int data_size[2];
__device__ float C;


__device__ float* get_element(int id, int set);
__device__ float dot(float* px, float *py)
{
//    print_vector(px);
//    print_vector(py);
	float sum = 0.0;
	int i;
	for(i=0; i< maximum_index; i++)
//	while(px->index != -1 && py->index != -1)
	{
		//printf(" i = %d  px = %f  py = %f  sum = %f \n", i, px[i], py[i], sum);
		sum += px[i] * py[i];
	}
	return sum;
}

inline float powi(float base, int times)
{
	float tmp = base, ret = 1.0;

    int t;
	for(t=times; t>0; t/=2)
	{
		if(t%2==1) ret*=tmp;
		tmp = tmp * tmp;
	}
	return ret;
}

//float kernel_linear(int set1, int element1, int set2, int element2) //todo: als template implementieren
__device__ float kernel(int set1, int element1, int set2, int element2)
{
//	g_data[set1][ element1 * max_vector[set1] ]
//    float ret = dot(prob[set1].x[element1], prob[set2].x[element2]);
	float* px = &(g_data[set1][ element1 * maximum_index ]);
	float* py = &(g_data[set2][ element2 * maximum_index ]);

	//int i;
	//for(i=0; i< maximum_index; i++)
		//printf("func kernel px: %f %f (index = %d)\n", px[i], g_data[set1][ element1 * maximum_index + i] ,  element1 * maximum_index + i);

	//for(i=0; i< maximum_index; i++)
		//printf("func kernel py: %f %f (index = %d)\n", py[i], g_data[set2][ element2 * maximum_index + i] ,  element2 * maximum_index + i);

	//printf(" dot: %d %d - %d %d \n", set1, element1, set2, element2);
    float ret = dot(px, py );
    if(set1 == set2 && element1 == element2)
        ret += C;
    return ret;
}

/*float kernel_poly(int set1, int element1, int set2, int element2)
{
    float ret = powi(param.gamma*dot(prob[set1].x[element1], prob[set2].x[element2])+param.coef0,param.degree);
    if(set1 == set2 && element1 == element2)
        ret += param.C;
    return ret;
}

float kernel_rbf(int set1, int element1, int set2, int element2)
{
    float dots = ( dot(prob[set1].x[element1], prob[set1].x[element1])+
                        dot(prob[set1].x[element1], prob[set2].x[element2])-2*
                        dot(prob[set1].x[element1], prob[set2].x[element2]));
    float wgamma = -param.gamma*dots;
    float wexp = exp(wgamma);

    if(set1 == set2 && element1 == element2)
        wexp += param.C;
    return wexp;

}

float kernel_sigmoid(int set1, int element1, int set2, int element2)
{
    float ret = tanh(param.gamma*dot(prob[set1].x[element1], prob[set2].x[element2])+param.coef0);
    if(set1 == set2 && element1 == element2)
        ret += param.C;
    return ret;
}
*/
/*float kernel_precomputed(int set1, int element1, int set2, int element2)
{
    return x[i][(int)(x[j][0].value)].value;
}*/

__device__ int find_max(int p, float *dot_yi_x, float* dot_xi_x, float dot_xi_yi, float dot_xi_xi, float *max_p) {
    // find max
    int max_p_index = -1;
    *max_p = -1000000000.0; //todo: HUGE_VAL fuer Cuda finden
    int i;
    for (i=0;i<data_size[p];i++) {
        float sum = dot_yi_x[i] - dot_xi_x[i] - dot_xi_yi + dot_xi_xi;
        //printf("sum:%f = dot_yi_x[%d]:%f - dot_xi_x[%d]:%f - dot_xi_yi:%f + dot_xi_xi:%f \n", sum, i, dot_yi_x[i], i,  dot_xi_x[i], dot_xi_yi, dot_xi_xi);
        if(sum > *max_p)
        {
            *max_p = sum;
            max_p_index = i;
        }
    }
    return max_p_index;
}


__device__ float compute_zaehler(float dot_xi_yi, float* dot_yi_x, float* dot_xi_x, int p, int max_p_index ) {
    float zaehler = dot_xi_yi - dot_yi_x[max_p_index] - dot_xi_x[max_p_index] + kernel(p,max_p_index, p, max_p_index); //todo: samevector, kann vorberechnet werden.
    return zaehler;
}

__device__ float compute_nenner(float dot_xi_xi, float* dot_xi_x, int p, int max_p_index) {
    float nenner = dot_xi_xi - 2* dot_xi_x[max_p_index] +  kernel(p, max_p_index, p, max_p_index);
    return nenner;
}

__device__ void add_to_weights(float* weights, float lambda, int index, int set)
{
    int i;
    for (i=0;i<data_size[set];i++)
    {
        if (i!=index)
            weights[i] *= lambda;
        else
            weights[i] = weights[i]*lambda + (1.0 - lambda)*1;
    }
}

__device__ float update_xi_xi(float dot_xi_xi, float* dot_xi_x, int p, int max_p_index, float lambda) {
    dot_xi_xi = lambda * lambda * dot_xi_xi
            + 2 * lambda * (1.0 - lambda) * dot_xi_x[max_p_index]
            + (1.0 - lambda)*(1.0 - lambda)*kernel(p, max_p_index, p ,max_p_index );
    return dot_xi_xi;
}

__device__ float update_xi_yi(float dot_xi_yi, float* dot_yi_x, int max_p_index, float lambda) {
    dot_xi_yi = lambda * dot_xi_yi + (1.0 - lambda) * dot_yi_x[max_p_index];
    return dot_xi_yi;
}

__device__ void update_xi_x(float* dot_xi_x, int p, int p2, int max_p_index, float lambda) {
    //printf("update_xi_x(): %d %d %d \n", p, p2, max_p_index);
    float* computed_kernels = get_element(max_p_index, p);

    int i;
    for (i=0;i<data_size[p2];i++) {
        //dot_xi_x[i]= dot_xi_x[i] * lambda + (1.0 - lambda) * kernel(p, max_p_index, p2, i);
        int offset = p2 * data_size[0];
        dot_xi_x[i]= dot_xi_x[i] * lambda + (1.0 - lambda) * computed_kernels[ offset + i  ]; //(p, max_p_index, p2, i);
        //printf(" %d - %f, max_p_index = %d, offset = %d\n", i, computed_kernels[ offset + i  ], max_p_index, offset);
    }
    //printf("\n");
}


// cache anfang

__device__ int nr_of_cache_entries;
__device__ int nr_of_elements;

__device__ float* data;

__device__ int* look_up_table; // translates data id to cache position
__device__ int* reverse_look_up_table; // translates cache positions to id
__device__ int* circular_array; // safes order in which cache pos was inserted

__device__ int ca_first;
__device__ int ca_last;
__device__ int ca_free_pos; // safes which pos has no yet been occupied

__device__ void get_data(int id, int set, int pointer)
{
//    data[pointer] = (double) id * id;

    int i;
    for(i=0;i<data_size[0];i++)
    {
        //printf("set1 = %d, id = %d,  set2 = %d, id = %d res = %f\n", set, id, 0, i,  kernel(set, id, 0, i));
        data[pointer * nr_of_elements + i] = kernel(set, id, 0, i);
    }

    for(i=0;i<data_size[1];i++)
    {
        //printf("set1 = %d, id = %d,  set2 = %d, id = %d   res = %f \n", set, id, 1, i,  kernel(set, id, 1, i));
        data[pointer * nr_of_elements + i + data_size[0]] = kernel(set, id, 1, i);
    }
}


__device__ void ca_add(int id) {
    int last_id = reverse_look_up_table[ circular_array[ca_last] ]; // clean up look up table
    if(circular_array[ca_last] != -1)
    {
        //pos = look_up_table[ last_id ];
        look_up_table[ last_id ] = -1;
    } else {
        circular_array[ca_last] = ca_free_pos;
        ca_free_pos++;
    }

    //circular_array[ca_last] = pos;
    ca_first = ca_last;
    ca_last = ca_last - 1;
    if(ca_last<0) ca_last = nr_of_cache_entries - 1;

    reverse_look_up_table[circular_array[ca_first]] = id;
    look_up_table[id] = circular_array[ca_first];
}

__device__ void ca_bring_forward(int pos)
{
//    printf("bring_fordward. enter. pos = %d\n", pos);
    int current = ca_first;
    int pos_temp = circular_array[current];
    int pos_temp2 = -1;
//    int i;
//    printf("circular array: ");
//    for(i=0; i< nr_of_cache_entries; i++)
//        printf(" %d: %d - ", i, circular_array[i]);
//    printf("\n");

//    printf("lut: ");
//    for(i=0; i< nr_of_elements; i++)
//        printf(" %d: %d - ", i, look_up_table[i]);
//    printf("\n");

//    printf("first = %d   last = %d \n", ca_first, ca_last);


    do{
//        printf("bring_fordward. cycle. \n");

        pos_temp2 = pos_temp;
        current = current + 1;
        if(current>=nr_of_cache_entries) current = 0;
        pos_temp = circular_array[current];
//        printf("current = %d   last = %d  pt = %d, pt2 = %d\n", current, last, pos_temp, pos_temp2);
        circular_array[current] = pos_temp2;

//    printf("circular array 2: ");
//    for(i=0; i< nr_of_cache_entries; i++)
//        printf(" %d: %d - ", i, circular_array[i]);
//    printf("\n");

    } while( pos_temp != pos);

    circular_array[ca_first] = pos;

//    printf("circular array 3: ");
//    for(i=0; i< nr_of_cache_entries; i++)
//        printf(" %d: %d - ", i, circular_array[i]);
//    printf("\n");

}

__device__ float* get_element(int id, int set)
{
    //printf(" get_element(): id = %d, set = %d \n", id, set);
    int idset = id + set* data_size[0];

	//printf("idset = %d \n", idset);

    if( look_up_table[idset] == -1 ) { // cache miss
        ca_add(idset);
        get_data(id, set, circular_array[ca_first]);
        //printf("cache miss, id = %d, set = %d\n", id, set);
    } else { //cache hit
        //printf("cache hit\n");
        if(look_up_table[idset] != circular_array[ca_first])
        {
            ca_bring_forward(look_up_table[idset]);
        }
    }
    //printf("get_element = data[%d]  ca_first = %d \n", circular_array[ca_first], ca_first);
    return &data[circular_array[ca_first] * nr_of_elements];
}

// cache ende



__global__ void
cuda_kernel( float* g_data0, float* g_data1 , int g_maximum_index, int g_data0_size, int g_data1_size, float* g_weights0, float* g_weights1 ,
float *dot_xi_x, float *dot_yi_x, float *dot_xi_y, float *dot_yi_y,
									int g_nr_of_cache_entries, int g_nr_of_elements,
									int *g_look_up_table, int* g_reverse_look_up_table, int* g_circular_array, float* g_data_cache) //todo: bessere namen fuer cache-variablen finden.
{

	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	int t_set;
	int t_element;

	if(tid <= g_data0_size)
		t_set = 0;
	else
		t_set = 1;

	t_element = tid - (1 - t_set) * g_data0_size;

if(tid < g_data0_size + g_data1_size)
{
if(tid == 0) {
	// cache initialisieren
	look_up_table = g_look_up_table;
	reverse_look_up_table = g_reverse_look_up_table;
	circular_array = g_circular_array;
	data = g_data_cache;

    nr_of_cache_entries = g_nr_of_cache_entries;
    nr_of_elements = g_nr_of_elements;

    // init pointer
    ca_first = 0;
    ca_last = nr_of_cache_entries - 1;

	//cache init ende

	//todo: C als parameter uebergeben
	C = 0.0;
	//todo: gleich die richtigen arrays senden
	g_data[0] = g_data0;
	g_data[1] = g_data1;

	//int i,j,k;
	//for(int i=0;i<2;i++)
	//for(int j=0;j<prob[i].l;j++)
	//for(int k=0;k<max_index;k++)
	//{
		//printf("on device:  i = %d,  j = %d  k = %d  value = %f ( index = %d )\n ", i, j, k, g_data[i][ max_index * j + k ], max_index * j + k );
	//}


	maximum_index = g_maximum_index;
	data_size[0] = g_data0_size;
	data_size[1] = g_data1_size;

	float* g_weights[2];
	g_weights[0] = g_weights0;
	g_weights[1] = g_weights1;
}

    // initialize weights  -- 0 == x, 1 == y
/*    int i;
    for (i=0;i<data_size[0];i++)
        g_weights[0][i] = 0.0;

    for (i=0;i<data_size[1];i++)
        g_weights[1][i] = 0.0;*/

	g_weights[t_set][t_id] = 0.0;

if(tid == 0) {
    g_weights[0][0] = 1.0;
    g_weights[1][0] = 1.0;


    // deklaration der variablen die werte zwischenspeichern
    //float *dot_xi_x; // < x_i, x> \forall x \in P
    //float *dot_yi_x;  // < y_i, x> \forall x \in P
    float dot_xi_yi; // <x_i, y_i >
    float dot_xi_xi; // <x_i, x_i >

    //float *dot_yi_y; // < y_i, y> \forall y \in Q
    //float *dot_xi_y;  // < x_i, y> \forall y \in Q
    float dot_yi_yi; // <y_i, y_i >


    // speicher anfordern 

    // initialisieren
    for (i=0;i<data_size[0];i++) {
        dot_xi_x[i]=kernel(0, 0, 0, i);
        dot_yi_x[i]=kernel(1, 0, 0, i);
    }

    for (i=0;i<data_size[1];i++) {
        dot_xi_y[i]=kernel(0, 0, 1, i);
        dot_yi_y[i]=kernel(1, 0, 1, i);
    }

    dot_xi_xi = kernel(0, 0, 0, 0);
    dot_xi_yi = kernel(0, 0, 1, 0);
    dot_yi_yi = kernel(1, 0, 1, 0);

    // find max
    int max_p_index;
    float max_p;
    max_p_index = find_max(0, dot_yi_x, dot_xi_x, dot_xi_yi, dot_xi_xi, &max_p);

    int max_q_index;
    float max_q;
    max_q_index = find_max(1, dot_xi_y, dot_yi_y, dot_xi_yi, dot_yi_yi, &max_q);

    int j;

    for (j=0;j<10 ;j++)
    {
        //printf("j = %d \n", j);
        float lambda;
        if (max_p >= max_q)
        {
            float zaehler = compute_zaehler(dot_xi_yi, dot_yi_x, dot_xi_x, 0, max_p_index);
            float nenner = compute_nenner(dot_xi_xi, dot_xi_x, 0, max_p_index);

            lambda = zaehler / nenner;

            if(zaehler == 0.0 && nenner == 0.0) lambda = 0.0;
            if(lambda < 0.0)	lambda = 0.0;
            if(lambda > 1.0)	lambda = 0.0;

            add_to_weights(g_weights[0], lambda, max_p_index, 0);

            // update dotproducts

            dot_xi_xi = update_xi_xi(dot_xi_xi, dot_xi_x, 0, max_p_index, lambda);

            dot_xi_yi = update_xi_yi(dot_xi_yi, dot_yi_x, max_p_index, lambda);

            //printf("max_p: \n");
            update_xi_x(dot_xi_x, 0, 0, max_p_index, lambda);

            update_xi_x(dot_xi_y, 0, 1, max_p_index, lambda);
        //printf("max_p = %f  max_q = %f zaehler = %f nenner = %f lambda = %f\n", max_p, max_q, zaehler, nenner, lambda);
        }
        else
        {
            double zaehler = compute_zaehler(dot_xi_yi, dot_xi_y, dot_yi_y, 1, max_q_index);
            double nenner = compute_nenner(dot_yi_yi, dot_yi_y, 1, max_q_index);

            lambda = zaehler / nenner;

            if(zaehler == 0.0 && nenner == 0.0) lambda = 0.0;
            if(lambda < 0.0)	lambda = 0.0;
            if(lambda > 1.0)	lambda = 0.0;

            add_to_weights(g_weights[1], lambda, max_q_index, 1);

            // update dotproducts

            dot_yi_yi = update_xi_xi(dot_yi_yi, dot_yi_y, 1, max_q_index, lambda);

            dot_xi_yi = update_xi_yi(dot_xi_yi, dot_xi_y, max_q_index, lambda);

            //printf("max_q: \n");
            update_xi_x(dot_yi_y, 1, 1, max_q_index, lambda);

            update_xi_x(dot_yi_x, 1, 0, max_q_index, lambda);
        //printf("max_p = %f  max_q = %f zaehler = %f nenner = %f lambda = %f\n", max_p, max_q, zaehler, nenner, lambda);
        }
        // find max
        max_p_index = find_max(0, dot_yi_x, dot_xi_x, dot_xi_yi, dot_xi_xi, &max_p);
        max_q_index = find_max(1, dot_xi_y, dot_yi_y, dot_xi_yi, dot_yi_yi, &max_q);
       //duality gap
        // absolute duality gap

        float adg = max_p + max_q;

        //printf("max_p = %f  max_q = %f ", max_p, max_q);
        //printf("adg = %f ", adg);

        // relative duality gap
        // adg / ||p-q||^2 - adg
        // adg / <p-q, p-q> - adg

        float distance = dot_xi_xi + dot_yi_yi - 2 * dot_xi_yi;


        float rdg_nenner = distance - adg;
        float rdg;

        if (rdg_nenner <= 0)
        {
            //printf("set huge value... ");
            rdg = 100000000000.0; // todo: HUGE_VAL;
        }
        else
        {
            rdg = adg / rdg_nenner;
        }

		//printf("<x-y,x-y> = %e " , distance);
		//printf("adg = %e " , adg);
        //printf("rdg = %e \n", rdg);
		//print_weights(x_weights, prob[0]);
		//print_weights(y_weights, prob[1]);

        //rho = - dot_xi_yi + dot_xi_xi - (dot_xi_xi + dot_yi_yi - 2 * dot_xi_yi)/2;
        float rho = dot_xi_yi - dot_xi_xi - (dot_xi_xi + dot_yi_yi - 2 * dot_xi_yi)/2;
        //printf("xi_xi = %f   yi_yi = %f   xi_yi = %f \n", dot_xi_xi, dot_yi_yi, dot_xi_yi);
	}
 	} 
	}
//	struct svm_problem d_prob[2];

//	int *test;
//	hipMalloc( (void**) &test, 100 * sizeof(int) );

//	int tid = threadIdx.x + blockDim.x*blockIdx.x;
//	g_data[tid] = tid; 
}

#endif // #ifndef _CUDA_KERNEL_H_
